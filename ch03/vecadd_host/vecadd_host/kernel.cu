#include "hip/hip_runtime.h"
﻿// PP_vectadd_host.cpp : 이 파일에는 'main' 함수가 포함됩니다. 거기서 프로그램 실행이 시작되고 종료됩니다.
//
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
//#include <iostream>



//c,a,b 배열은 device에 선언되어 있어야 한다.
__global__ void addKernel(int* c, const int* a, const int* b) {
	int i = threadIdx.x;//어느 core가 실행중인지가 (몇번쨰 core가) x에 담김
	c[i] = a[i] + b[i];
}

int main(void) {
	const int SIZE = 5;
	const int a[SIZE] = { 1,2,3,4,5 };
	const int b[SIZE] = { 10,20,30,40,50 };
	int c[SIZE] = { 0 };

	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	//allocate device memory
	hipMalloc((void**)&dev_a, SIZE * sizeof(int));
	hipMalloc((void**)&dev_b, SIZE * sizeof(int));
	hipMalloc((void**)&dev_c, SIZE * sizeof(int));

	//copy from host to device
	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

	//lauch a kernel on the GPU with one thread for each element
	addKernel << <1, SIZE >> > (dev_c, dev_a, dev_b);


	//copy from device to host
	hipMemcpy(c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	//free device memory
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	//print the result
	printf("{%d, %d, %d,%d,%d}+{%d, %d, %d,%d,%d}""={%d, %d, %d,%d,%d}\n",
		a[0], a[1], a[2], a[3], a[4],
		b[0], b[1], b[2], b[3], b[4],
		c[0], c[1], c[2], c[3], c[4]);
	return 0;
}
