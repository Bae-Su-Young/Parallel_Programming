#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


#define GRIDSIZE (32*1024)
#define BLOCKSIZE 1024
#define TOTALSIZE (GRIDSIZE*BLOCKSIZE)

__global__ void kernel(unsigned long long int* pCount) {
	(*pCount) = (*pCount) + 1;
}

//atomic version
__global__ void kernel_atomic(unsigned long long int* pCount) {
	atomicAdd(pCount, 1ULL);
}

__global__ void kernel_atomic_sharedMem(unsigned long long int* pCount) {
	__shared__ int nCountShared;		
	if (threadIdx.x == 0) {
		nCountShared = 0;
	}

	__syncthreads();
	atomicAdd(&nCountShared, 1);		//atomic add ����
	__syncthreads();

	if (threadIdx.x == 0) {				//shared memory�� �ִ� ����
		atomicAdd(pCount, nCountShared);//�ٷ� global memory�� ����
	}

}
int main(void) {
	unsigned long long int aCount[1];

	//prepare timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//cuda: alocate device memory
	unsigned long long int* pCountDev = NULL;
	hipMalloc((void**)&pCountDev, sizeof(unsigned long long int));
	hipMemset(pCountDev, 0, sizeof(unsigned long long int));

	//start timer
	hipEventRecord(start, 0);

	//launch kernel
	dim3 dimGrid(GRIDSIZE, 1, 1);
	dim3 dimBlock(BLOCKSIZE, 1, 1);
	//kernel << <dimGrid, dimBlock >> > (pCountDev);
	//kernel_atomic << <dimGrid, dimBlock >> > (pCountDev);
	kernel_atomic_sharedMem << <dimGrid, dimBlock >> > (pCountDev);

	//copy from device to host
	hipMemcpy(aCount, pCountDev, sizeof(unsigned long long int), hipMemcpyDeviceToHost);

	printf("total number of threads=%d\n", TOTALSIZE);
	printf("count = % llu\n", aCount[0]);

	//end timer
	float time;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("elpased time= %f msec\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	//hipFree
	hipFree(pCountDev);


}