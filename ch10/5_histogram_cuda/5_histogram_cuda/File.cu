#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <windows.h> 
#include "hip/hip_runtime.h"
#include ""


#define GRIDSIZE 1024
#define BLOCKSIZE 1024
#define TOTALSIZE (GRIDSIZE*BLOCKSIZE)

#define NUMHIST 16 

void genData(unsigned int* ptr, unsigned int size) {
	while (size--) {
		*ptr++ = (unsigned int)(rand() % (NUMHIST - 1));
	}
}

__global__ void kernel(unsigned int* hist, unsigned int* img, unsigned int size) {

	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int pixelVal = img[i];
	atomicAdd(&(hist[pixelVal]), 1);

}
//sharedm memory version
__global__ void kernel_sharedMem(unsigned int* hist, unsigned int* img, unsigned int size) {
	__shared__ int histShared[NUMHIST];
	if (threadIdx.x < NUMHIST) {		//ó�� 16������� shared memory initialize
		histShared[threadIdx.x] = 0;
	}
	__syncthreads();
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int pixelVal = img[i];
	atomicAdd(&(histShared[pixelVal]), 1);
	__syncthreads();

	if (threadIdx.x < NUMHIST) {
		atomicAdd(&(hist[threadIdx.x]), histShared[threadIdx.x]); //global memory�� �������� ���ϴ� ����
	}
}

int main(void) {
	unsigned int* pImage = NULL;
	unsigned int* pHistogram = NULL;
	int i;

	//prepare timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//malloc
	pImage = (unsigned int*)malloc(TOTALSIZE * sizeof(unsigned int));
	pHistogram = (unsigned int*)malloc(NUMHIST * sizeof(unsigned int));
	for (i = 0; i < NUMHIST; i++) {
		pHistogram[i] = 0;
	}

	//generate src data
	genData(pImage, TOTALSIZE);

	//CUAD: allocate device memory
	unsigned int* pImageDev;
	unsigned int* pHistogramDev;
	hipMalloc((void**)&pImageDev, TOTALSIZE * sizeof(unsigned int));
	hipMalloc((void**)&pHistogramDev, NUMHIST * sizeof(unsigned int));
	hipMemset(pHistogramDev, 0, NUMHIST * sizeof(unsigned int));
	
	//CUDA: copy from host to side
	hipMemcpy(pImageDev, pImage, TOTALSIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

	//start timer
	hipEventRecord(start, 0);

	//kernel(pHistogram, pImage, TOTALSIZE);
	dim3 dimGrid(GRIDSIZE, 1, 1);
	dim3 dimBlock(BLOCKSIZE, 1, 1);
	//kernel << <dimGrid, dimBlock >> > (pHistogramDev, pImageDev, TOTALSIZE);
	kernel_sharedMem << <dimGrid, dimBlock >> > (pHistogramDev, pImageDev, TOTALSIZE);
	//end timer
	float time;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("elapsed time=%f msec\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//CUDA: copy from device to host
	hipMemcpy(pHistogram, pHistogramDev, NUMHIST * sizeof(unsigned int), hipMemcpyDeviceToHost);

	//rpitn histogram
	long total = 0L;
	for (i = 0; i < NUMHIST; i++) {
		printf("%2d: %10d\n", i, pHistogram[i]);
		total += pHistogram[i];
	}
	printf("total: %10ld (should be %ld)\n", total, TOTALSIZE);
}